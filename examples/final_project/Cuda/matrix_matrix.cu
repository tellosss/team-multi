#include "hip/hip_runtime.h"
// =================================================================
//
// File: matrix_matrix.cu
// Author: Isaac Planter Villalobos A01702962
// Description: This file implements the multiplication of a matrix
//				by another matrix using CUDA.
//
// Copyright (c) 2021 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define RENS    3
#define COLS    3
#define THREADS 256
#define BLOCKS	MMIN(32, (((RENS * COLS) / THREADS) + 1))

__global__ void matrix_matrix(int *m1, int *m2, int *c) {
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int i, j, acum, pos=0;


    while (tid < RENS){
        for (i = 0; i < COLS; i++) {
            acum = 0;
            for (j = 0; j < RENS; j++) {
                acum += (m1[(tid * COLS) + j] * m2[(j*RENS)+i]);
                // acum += (m1[(i * COLS) + j] * m2[(j*COLS)+i]);

            }
            // c[pos] = acum;
            c[tid*COLS)+i]= acum;
            pos++;
            
        }
        tid += blockDim.x * gridDim.x;
    }
}

int main(int argc, char* argv[]) {
	int i, j, *m1, *m2, *c;
    int *d_m1, *d_m2, *d_c;
	double ms;

	m1 = (int*) malloc(sizeof(int) * RENS* COLS);
	m2 = (int*) malloc(sizeof(int) * RENS* COLS);
	c = (int*) malloc(sizeof(int) * RENS* COLS);

    for (i = 0; i < RENS; i++) {
        for (j = 0; j < COLS; j++) {
            m1[(i * COLS) + j] = (j + 1);
            m2[(i * COLS) + j] = (j + 1);
        }
    }

    hipMalloc((void**)&d_m1, sizeof(int) * RENS* COLS);
    hipMalloc((void**)&d_m2, sizeof(int) * RENS* COLS);
    hipMalloc((void**)&d_c, sizeof(int) * RENS* COLS);

    hipMemcpy(d_m1, m1, sizeof(int) * RENS* COLS, hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2, sizeof(int) * RENS* COLS, hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();

		matrix_matrix<<<BLOCKS, THREADS>>>(d_m1, d_m2, d_c);

		ms += stop_timer();
	}

    hipMemcpy(c, d_c, sizeof(int) * RENS, hipMemcpyDeviceToHost);

	display_array("c:", c);
	printf("avg time = %.5lf ms\n", (ms / N));

    hipFree(d_m1); hipFree(d_m2); hipFree(d_c);
	free(m1); free(m2); free(c);
	return 0;
}
