#include "hip/hip_runtime.h"
// =================================================================
//
// File: example8.cu
// Author(s): Isaac Planter A01702962 Sandra Tello A01703658
// Speedup achieved: 55193.38462
// Description: This file contains the code that implements the
//				enumeration sort algorithm using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 10000
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void enumerationSort(int*arr, int*aux, int size){

   int tid = threadIdx.x + (blockIdx.x * blockDim.x);

   while (tid < size) {
		int menores = 0;
		for (int j = 0; j< size; j++){
			if(arr[tid]>arr[j] || (arr[tid]==arr[j]&&tid<j) ){
				menores+=1;
			}
		}
		aux[menores] = arr[tid];
        tid += blockDim.x * gridDim.x;
	}
	for (int k = 0; k<size; k++){
		arr[k] = aux[k];
	}
}


int main(int argc, char* argv[]) {
	int i, j, *a, *aux, *d_a, *d_aux;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
    aux = (int *) malloc(sizeof(int) * SIZE);
    random_array(a, SIZE);
	display_array("before", a);
    
    hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
    hipMalloc( (void**) &d_aux, SIZE * sizeof(int) );

    printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
        hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_aux, aux, SIZE * sizeof(int), hipMemcpyHostToDevice);

	    start_timer();

        enumerationSort<<<1, THREADS>>>(d_a, d_aux, SIZE);
        // for (int k = 0; k<SIZE; k++){
		// 	a[k] = aux[k];
		// }

	    ms += stop_timer();
	}

    hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(aux, d_aux, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    printf("avg time = %.5lf ms\n", (ms / N));

    display_array("after", a);


	hipFree(d_a);
    hipFree(d_aux);

	free(a);
    free(aux);

  return 0;
}
