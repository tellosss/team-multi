#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Author(s):
// Description: This file contains the code to count the number of
//				even numbers within an array using CUDA.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__device__ int NumberOfEvens(int num) {
    if(num % 2 == 0)
	    return 1;
    else
        return 0;
}

__global__ void NumberOFEvens(int *array, int *results) {
	__shared__ int cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

	int aux = 0;
	while (tid < SIZE) {
		//aux = (aux < array[tid])? aux : array[tid];
		aux = aux + NumberOfEvens(array[tid]);
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = aux;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] = NumberOfEvens(cache[cacheIndex], cache[cacheIndex + 1]);
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		results[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	int i, *a, *results, pos;
  	int *d_a, *d_r;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
	fill_array(a, SIZE);
	display_array("a", a);

	srand(time(0));
	pos = rand() % SIZE;
	printf("Setting value 0 at %i\n", pos);
	a[pos] = 0;

  	results = (int *) malloc(sizeof(int) * BLOCKS);

	hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(int) );

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();

		NumberOfEvens<<<BLOCKS, THREADS>>> (d_a, d_r);

		ms += stop_timer();
	}

	hipMemcpy(results, d_r, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

	int aux = 0;
	for (i = 0; i < BLOCKS; i++) {
		aux = aux + results[i];
	}

	printf("Number of Evens = %i\n", aux);
	printf("avg time = %.5lf\n", (ms / N));

	hipFree(d_r);
	hipFree(d_a);

	free(a);
  	free(results);
	return 0;
}
// implement your code
