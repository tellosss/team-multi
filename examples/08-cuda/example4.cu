#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Author: Isaac Planter A01702962 Sandra Tello A01703658
// Description: This file contains the code to count the number of
//				even numbers within an array using using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000 //1e9
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void NumberOfEvens(int *array, long *result) {
	__shared__ long cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

	long acum = 0;
	while (tid < SIZE) {
        if(array[tid] % 2 == 0)
			acum++;
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = acum;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		result[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	int i, *array, *d_a;
	long *results, *d_r;
	double ms;

	array = (int*) malloc( SIZE * sizeof(int) );
	fill_array(array, SIZE);
	display_array("array", array);

	results = (long*) malloc( BLOCKS * sizeof(long) );

	hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(long) );

	hipMemcpy(d_a, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();

		NumberOfEvens<<<BLOCKS, THREADS>>> (d_a, d_r);

		ms += stop_timer();
	}

	hipMemcpy(results, d_r, BLOCKS * sizeof(long), hipMemcpyDeviceToHost);

	long acum = 0;
	for (i = 0; i < BLOCKS; i++) {
		acum += results[i];
	}

	printf("result = %li\n", acum);
	printf("avg time = %.5lf\n", (ms / N));

	hipFree(d_r);
	hipFree(d_a);

	free(array);
	free(results);
	return 0;
}
