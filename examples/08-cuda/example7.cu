#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Author(s):
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000 //1e6
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void ChecaPrimos(int*arr, int size){
   int i, aux, RaizCuadrada;
   int tid = threadIdx.x + (blockIdx.x * blockDim.x);
   RaizCuadrada = 0;

   while (tid < size) {
       RaizCuadrada = sqrtf(tid);
		bool Bandera = false;
		for(int j=2; j<=RaizCuadrada; j++){
			if(tid % j==0){
				//No es primo
				Bandera = true;
				break;
			}
		}
		if(Bandera==false){
			arr[tid]=1;
		}
		tid += blockDim.x * gridDim.x;
	}

}

// __global__ void even(int* arr, int size) {
//   int i, aux, RaizCuadrada;
//   RaizCuadrada = 0;

//   i = (threadIdx.x * 2);
//   if (i <= size - 2) {
//     if (arr[i] > arr[i + 1]) {
//       aux = arr[i];
//       arr[i] = arr[i + 1];
//       arr[i + 1] = aux;
//     }
//   }
// }

// __global__ void odd(int* arr, int size) {
//   int i, aux;

//   i = (threadIdx.x * 2) + 1;
//   if (i <= size - 2) {
//     if (arr[i] > arr[i + 1]) {
//       aux = arr[i];
//       arr[i] = arr[i + 1];
//       arr[i + 1] = aux;
//     }
//   }
// }

int main(int argc, char* argv[]) {
	int i, j, *a, *d_a;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
    printf("At first, neither is a prime. We will display to TOP_VALUE:\n");
	for (i = 2; i < TOP_VALUE; i++) {
		if (a[i] == 0) {
			printf("%i ", i);
		}
	}
    printf("\n");


    hipMalloc( (void**) &d_a, SIZE * sizeof(int) );

    printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
        hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

	    start_timer();

        ChecaPrimos<<<1, THREADS>>>(d_a, SIZE);

	    ms += stop_timer();
	}

    hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    printf("avg time = %.5lf ms\n", (ms / N));

    display_array("array", a);

    for (i = 2; i < TOP_VALUE; i++) {
		if (a[i] == 1) {
			printf("%i ", i);
		}
	}

	hipFree(d_a);

	free(a);

  return 0;
}
